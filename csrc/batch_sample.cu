#include "hip/hip_runtime.h"
/*
    sample in batch
*/


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <pybind11/pybind11.h>

namespace py = pybind11;

/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOTAL_THREADS_DENSE 512

inline int opt_n_threads(int work_size)
{
    const int pow_2 = std::log(static_cast<double>(work_size)) / std::log(2.0);

    return max(min(1 << pow_2, TOTAL_THREADS_DENSE), 1);
}

#define CUDA_CHECK_ERRORS()                                                                        \
    do                                                                                             \
    {                                                                                              \
        hipError_t err = hipGetLastError();                                                      \
        if (hipSuccess != err)                                                                    \
        {                                                                                          \
            fprintf(stderr, "CUDA kernel failed : %s\n%s at L:%d in %s\n",                         \
                    hipGetErrorString(err), __PRETTY_FUNCTION__, __LINE__, __FILE__);             \
            exit(-1);                                                                              \
        }                                                                                          \
    } while (0)

#define CHECK_CONTIGUOUS(x)                                                                        \
    do                                                                                             \
    {                                                                                              \
        TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor");                         \
    } while (0)

#define CHECK_IS_FLOAT(x)                                                                          \
    do                                                                                             \
    {                                                                                              \
        TORCH_CHECK(x.scalar_type() == at::ScalarType::Float, #x " must be a float tensor");       \
    } while (0)

#define CHECK_CUDA(x)                                                                              \
    do                                                                                             \
    {                                                                                              \
        TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor");                                     \
    } while (0)

#ifdef VERSION_GE_1_3
#define DATA_PTR data_ptr
#else
#define DATA_PTR data
#endif


#define MAX2(x, y) ((x)>(y) ? (x): (y))
#define MAX3(x, y, z) (MAX2(MAX2(x,y), z))

/////////////////////////////////////////////////////////////////////////////////////////////////////////////





torch::Tensor batch_sample(torch::Tensor batch, torch::Tensor num, torch::Tensor rand_num, torch::Tensor start_end_ind)
{

    // sort
    auto first = thrust::make_zip_iterator(thrust::make_tuple(segments.begin(), key_vec.begin()));
    thrust::stable_sort_by_key(value_ptr, value_ptr + one_batch_len * batch_num, first, thrust::greater<FloatType>());

    // NOTE: thrust of old version may fail to execute this command (and compile with warnings)
    thrust::stable_sort_by_key(segments.begin(), segments.begin() + one_batch_len * batch_num, key_vec.begin(), thrust::less<int>());
}