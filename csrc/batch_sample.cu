/*
    sample in batch
*/


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <pybind11/pybind11.h>

namespace py = pybind11;

/////////////////////////////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////////////////////////////////////////


torch::Tensor batch_sample(torch::Tensor batch, torch::Tensor num, torch::Tensor rand_num, torch::Tensor start_end_ind)
{

    // sort
    auto first = thrust::make_zip_iterator(thrust::make_tuple(segments.begin(), key_vec.begin()));
    thrust::stable_sort_by_key(value_ptr, value_ptr + one_batch_len * batch_num, first, thrust::greater<FloatType>());

    // NOTE: thrust of old version may fail to execute this command (and compile with warnings)
    thrust::stable_sort_by_key(segments.begin(), segments.begin() + one_batch_len * batch_num, key_vec.begin(), thrust::less<int>());
}



